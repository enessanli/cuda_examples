#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1); \
        } \
    }

__global__ 
void vec_add_device(float *a, float *b, float *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
        printf("GPU: c[%d] = %f (a[%d] = %f, b[%d] = %f)\n", index, c[index], index, a[index], index, b[index]);
    }
}

void vec_add_host(float *a, float *b, float *c, int n) {
    hipDeviceReset();

    float *d_a, *d_b, *d_c;
    int size = n * sizeof(float);
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    printf("CUDA Bellek: %ld MB boş, %ld MB toplam\n", free_mem / (1024 * 1024), total_mem / (1024 * 1024));

    CUDA_CHECK(hipMalloc((void **)&d_a, size));
    CUDA_CHECK(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMalloc((void **)&d_b, size));
    CUDA_CHECK(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMalloc((void **)&d_c, size));

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    vec_add_device<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        printf("host c[%d] = %f\n", i, c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    float a[1000], b[1000], c[1000];
    int n = 1000;
    
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = (3 * i * i);
    }
    
    vec_add_host(a, b, c, n);
}
